#include <iostream>
#include <chrono>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
using namespace std;

// 自定义 atomicAdd 支持 double 类型
__device__ double atomicAddDouble(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
            __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

// 策略B：一个Block负责一行，线程负责列元素
__global__ void division_kernel_row_col(double* A, double* b, int k, int n) {
    int row = k + 1 + blockIdx.x;
    int col = k + 1 + threadIdx.x;

    if (row < n && col < n) {
        double Aik = A[row * n + k];
        double Akk = A[k * n + k];
        A[row * n + col] -= Aik * A[k * n + col] / Akk;
    }

    __syncthreads();

    if (row < n && threadIdx.x == 0) {
        b[row] -= A[row * n + k] * b[k] / A[k * n + k];
        A[row * n + k] = 0.0;
    }
}

// 回代阶段：每次求解 x[k]
__global__ void back_substitution_kernel(double* A, double* b, double* x, int k, int n) {
    int j = threadIdx.x;

    if (j > k && j < n) {
        atomicAddDouble(&b[k], -A[k * n + j] * x[j]);
    }

    __syncthreads();

    if (j == 0) {
        x[k] = b[k] / A[k * n + k];
    }
}

// 初始化矩阵
void initialize_matrix(double* A, double* b, int n) {
    srand(time(0));
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            A[i * n + j] = rand() % 100 + 1;
        }
        b[i] = rand() % 100 + 1;
    }
}

int main() {
    int n;
    cout << "输入矩阵维度: ";
    cin >> n;

    double* A = new double[n * n];
    double* b = new double[n];
    double* x = new double[n];

    initialize_matrix(A, b, n);

    double *d_A, *d_b, *d_x;
    hipMalloc(&d_A, sizeof(double) * n * n);
    hipMalloc(&d_b, sizeof(double) * n);
    hipMalloc(&d_x, sizeof(double) * n);

    hipMemcpy(d_A, A, sizeof(double) * n * n, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(double) * n, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float elapsedTime = 0.0;

    hipEventRecord(start, 0);

    // 前向消元
    for (int k = 0; k < n - 1; ++k) {
        dim3 threadsPerBlock(256);
        dim3 blocksPerGrid(n - k - 1);
        division_kernel_row_col<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_b, k, n);
        hipDeviceSynchronize();
    }

    // 回代求解（倒序）
    for (int k = n - 1; k >= 0; --k) {
        int threads = (n > 1024) ? 1024 : n; // 最大线程数限制
        back_substitution_kernel<<<1, threads>>>(d_A, d_b, d_x, k, n);
        hipDeviceSynchronize();
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    cout << "[策略B + GPU回代] 完整GPU高斯消元完成，用时：" << elapsedTime << " ms" << endl;

    hipMemcpy(x, d_x, sizeof(double) * n, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_x);
    delete[] A;
    delete[] b;
    delete[] x;
    return 0;
}
