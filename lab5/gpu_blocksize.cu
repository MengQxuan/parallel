// #include <iostream>
// #include <chrono>
// #include <cstdlib>
// #include <ctime>
// #include <cuda_runtime.h>
// using namespace std;

// // 除法核函数  每个线程处理一整行
// __global__ void division_kernel(double* A, double* b, int k, int n) {
//     int tid = blockIdx.x * blockDim.x + threadIdx.x;
//     if (tid > k && tid < n) {
//         double factor = A[tid * n + k] / A[k * n + k];
//         for (int j = k + 1; j < n; ++j) {
//             A[tid * n + j] -= factor * A[k * n + j];
//         }
//         b[tid] -= factor * b[k];
//         A[tid * n + k] = 0.0;
//     }
// }

// // 初始化矩阵
// void initialize_matrix(double* A, double* b, int n) {
//     srand(time(0));
//     for (int i = 0; i < n; ++i) {
//         for (int j = 0; j < n; ++j) {
//             A[i * n + j] = rand() % 100 + 1;
//         }
//         b[i] = rand() % 100 + 1;
//     }
// }

// int main() {
//     int n;
//     cout << "输入矩阵维度: ";
//     cin >> n;

//     // 尝试不同的 block size 配置
//     int block_sizes[] = {64, 128, 256, 512, 1024};
//     int num_tests = sizeof(block_sizes) / sizeof(int);

//     for (int t = 0; t < num_tests; ++t) {
//         int threadsPerBlock = block_sizes[t];

//         // 主机内存分配
//         double* A = new double[n * n];
//         double* b = new double[n];
//         initialize_matrix(A, b, n);

//         // 设备内存分配
//         double *d_A, *d_b;
//         cudaMalloc(&d_A, sizeof(double) * n * n);
//         cudaMalloc(&d_b, sizeof(double) * n);

//         cudaMemcpy(d_A, A, sizeof(double) * n * n, cudaMemcpyHostToDevice);
//         cudaMemcpy(d_b, b, sizeof(double) * n, cudaMemcpyHostToDevice);

//         // 初始化计时器
//         cudaEvent_t start, stop;
//         cudaEventCreate(&start);
//         cudaEventCreate(&stop);
//         float elapsedTime = 0.0;

//         cudaEventRecord(start, 0);

//         // GPU 高斯消元主循环
//         for (int k = 0; k < n - 1; ++k) {
//             int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
//             division_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_b, k, n);
//             cudaDeviceSynchronize();
//         }

//         cudaEventRecord(stop, 0);
//         cudaEventSynchronize(stop);
//         cudaEventElapsedTime(&elapsedTime, start, stop);

//         cout << "[block size = " << threadsPerBlock << "] GPU高斯消元用时：" << elapsedTime << " ms" << endl;

//         // 释放资源
//         cudaEventDestroy(start);
//         cudaEventDestroy(stop);
//         cudaFree(d_A);
//         cudaFree(d_b);
//         delete[] A;
//         delete[] b;
//     }

//     return 0;
// }

#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
using namespace std;

// 策略B：每个Block处理一行，每个线程处理该行的一列
__global__ void division_kernel_row_col(double *A, double *b, int k, int n)
{
    int row = k + 1 + blockIdx.x;
    int col = k + 1 + threadIdx.x;

    if (row < n && col < n)
    {
        double Aik = A[row * n + k];
        double Akk = A[k * n + k];
        A[row * n + col] -= Aik * A[k * n + col] / Akk;
    }

    __syncthreads(); // 块内同步

    if (row < n && threadIdx.x == 0)
    {
        b[row] -= A[row * n + k] * b[k] / A[k * n + k];
        A[row * n + k] = 0.0;
    }
}

// 初始化矩阵
void initialize_matrix(double *A, double *b, int n)
{
    srand(time(0));
    for (int i = 0; i < n; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            A[i * n + j] = rand() % 100 + 1;
        }
        b[i] = rand() % 100 + 1;
    }
}

// 反向替代
void back_substitution(double *A, double *b, double *x, int n)
{
    x[n - 1] = b[n - 1] / A[n * n - 1];
    for (int i = n - 2; i >= 0; --i)
    {
        double sum = b[i];
        for (int j = i + 1; j < n; ++j)
        {
            sum -= A[i * n + j] * x[j];
        }
        x[i] = sum / A[i * n + i];
    }
}

int main()
{
    int n;
    cout << "输入矩阵维度: ";
    cin >> n;

    int block_sizes[] = {64, 128, 256, 512, 1024};
    int num_tests = sizeof(block_sizes) / sizeof(int);

    for (int t = 0; t < num_tests; ++t)
    {
        int threadsPerBlock = block_sizes[t];

        // 主机内存
        double *A = new double[n * n];
        double *b = new double[n];
        double *x = new double[n];
        initialize_matrix(A, b, n);

        // 设备内存
        double *d_A, *d_b;
        hipMalloc(&d_A, sizeof(double) * n * n);
        hipMalloc(&d_b, sizeof(double) * n);
        hipMemcpy(d_A, A, sizeof(double) * n * n, hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, sizeof(double) * n, hipMemcpyHostToDevice);

        // 计时器
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        float elapsedTime = 0.0;
        hipEventRecord(start, 0);

        // 消元主循环
        for (int k = 0; k < n - 1; ++k)
        {
            int needed_cols = n - (k + 1); // 每行剩余列数
            int actual_threads = min(threadsPerBlock, needed_cols);
            dim3 blockDim(actual_threads);
            dim3 gridDim(n - k - 1); // 每行一个 block
            division_kernel_row_col<<<gridDim, blockDim>>>(d_A, d_b, k, n);
            hipDeviceSynchronize();
        }

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop);
        cout << "[block size = " << threadsPerBlock << "] GPU高斯消元用时：" << elapsedTime << " ms" << endl;

        // 反代
        hipMemcpy(A, d_A, sizeof(double) * n * n, hipMemcpyDeviceToHost);
        hipMemcpy(b, d_b, sizeof(double) * n, hipMemcpyDeviceToHost);
        back_substitution(A, b, x, n);

        // 清理资源
        hipFree(d_A);
        hipFree(d_b);
        delete[] A;
        delete[] b;
        delete[] x;
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }
    return 0;
}
