#include <iostream>
#include <chrono>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
using namespace std;
using namespace std::chrono;

// 除法核函数
__global__ void division_kernel(double *A, double *b, int k, int n)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid > k && tid < n)
    {
        double factor = A[tid * n + k] / A[k * n + k];
        for (int j = k + 1; j < n; ++j)
        {
            A[tid * n + j] -= factor * A[k * n + j];
        }
        b[tid] -= factor * b[k];
        A[tid * n + k] = 0.0;
    }
}

// 初始化矩阵
void initialize_matrix(double *A, double *b, int n)
{
    srand(time(0));
    for (int i = 0; i < n; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            A[i * n + j] = rand() % 100 + 1;
        }
        b[i] = rand() % 100 + 1;
    }
}

// 反向替代（在 CPU 上执行）
void back_substitution(double *A, double *b, double *x, int n)
{
    x[n - 1] = b[n - 1] / A[n * n - 1];
    for (int i = n - 2; i >= 0; --i)
    {
        double sum = b[i];
        for (int j = i + 1; j < n; ++j)
        {
            sum -= A[i * n + j] * x[j];
        }
        x[i] = sum / A[i * n + i];
    }
}

int main()
{
    int n;
    cout << "输入矩阵维度: ";
    cin >> n;

    // 使用一维数组表示 A[n][n]
    double *A = new double[n * n];
    double *b = new double[n];
    double *x = new double[n];

    initialize_matrix(A, b, n);

    double *d_A, *d_b;
    hipMalloc(&d_A, sizeof(double) * n * n);
    hipMalloc(&d_b, sizeof(double) * n);

    hipMemcpy(d_A, A, sizeof(double) * n * n, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(double) * n, hipMemcpyHostToDevice);

    auto start = high_resolution_clock::now();

    for (int k = 0; k < n - 1; ++k)
    {
        int threadsPerBlock = 256;
        int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
        division_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_b, k, n);
        hipDeviceSynchronize();
    }

    // 把 A 和 b 拷贝回 CPU 执行回代
    hipMemcpy(A, d_A, sizeof(double) * n * n, hipMemcpyDeviceToHost);
    hipMemcpy(b, d_b, sizeof(double) * n, hipMemcpyDeviceToHost);

    back_substitution(A, b, x, n);

    auto end = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(end - start);
    cout << "GPU高斯消元完成，用时：" << duration.count() << "us" << endl;

    hipFree(d_A);
    hipFree(d_b);
    delete[] A;
    delete[] b;
    delete[] x;

    return 0;
}
